#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "GL/glm/glm.hpp"
#include "GL/glm/gtc/matrix_transform.hpp"
#include "GL/glm/gtc/matrix_access.hpp"
#include "GL/glm/gtc/matrix_inverse.hpp"
#include "GL/glm/gtc/type_ptr.hpp"

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
//#include <helper_gl.h>
//#if defined (__APPLE__) || defined(MACOSX)
//  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
//  #include <GLUT/glut.h>
//  #ifndef glutCloseFunc
//  #define glutCloseFunc glutWMCloseFunc
//  #endif
//#else
//#include <GL/freeglut.h>
//#endif

// Alternative, newer graphics libraries (compared to freeglut)
#include "GL/glew.h"
#include "GL/glfw3.h"

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <hip/hip_vector_types.h>
#include "polar.h"
#include "shaderWrapper.h"

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
GLFWwindow* window;
const unsigned int window_width  = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width    = 256;
const unsigned int mesh_height   = 256;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = 1.0;

StopWatchInterface *timer = NULL;

// glfw mouse controls
double glfw_mouse_old_y, glfw_mouse_old_x;

// Camera coordinates
glm::vec3 g_eyePosPolar(translate_z, rotate_x, rotate_y);
glm::vec3 g_eyePosCartesian(polarToCartesianPoint(g_eyePosPolar));
//glm::vec3 g_eyePosCartesian(0, 0, 1.0);
// Vectors for view matrix
glm::vec3 g_look_at_point(0.0, 0.0, 0.0);
glm::vec3 g_up_vector(0.0, 1.0, 0.0);
// View matrix
glm::mat4 g_viewMTX = glm::lookAt(g_eyePosCartesian, g_look_at_point, g_up_vector);
// Model matrix initialized as identity
glm::mat4 g_modelMTX(1.0f);
glm::mat4 g_projMTX = glm::perspective((GLfloat)glm::radians(60.0f), (GLfloat)1.0, (GLfloat)(0.1), (GLfloat)(300));
// Enums
GLint g_ProgramID;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void runAutoTest(int devID, char **argv, char *ref_file);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

const char *sSDKsample = "simpleGL (VBO)";

// GLFW functions
void glfw_keyboard(GLFWwindow* window, int key, int scancode, int action, int mods);
void glfw_mouse(GLFWwindow* window, int button, int action, int mods);
void glfw_cursor_pos_callback(GLFWwindow* window, double xpos, double ypos);
void glfw_display();
bool glfw_runTest(int argc, char** argv, char* ref_file);

void init_shaders()
{
    CShader myShaderWrap(".\\v_shader.glsl", ".\\f_shader.glsl");

    g_ProgramID = myShaderWrap.getProgram();

    myShaderWrap.use();
}

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

    // write output vertex
    pos[y*width+x] = make_float4(u, w, v, 1.0f);
}


void launch_kernel(float4 *pos, unsigned int mesh_width,
                   unsigned int mesh_height, float time)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel<<< grid, block>>>(pos, mesh_width, mesh_height, time);
}

struct point3 {
    GLfloat x;
    GLfloat y;
    GLfloat z;
};

struct point4 {
    GLfloat x;
    GLfloat y;
    GLfloat z;
    GLfloat w;
};

struct vertatt {
    point3 vertex;
    point4 color;
};

vertatt triangle[3] = {
    {-1.0, -1.0, 0.0, 1.0, 0, 0, 1.0},
    {1.0, -1.0, 0.0, 1.0, 0, 0, 1.0},
    {0, 1.0, 0.0, 1.0, 0, 0, 1.0}
};

GLuint tri_vao;
GLuint tri_vbo;

void basic_triangle_vbo() {
    glGenVertexArrays(1, &tri_vao);
    glBindVertexArray(tri_vao);

    glGenBuffers(1, &tri_vbo);
    glBindBuffer(GL_ARRAY_BUFFER, tri_vbo);
    glBufferData(GL_ARRAY_BUFFER, 3 * 7 * sizeof(GLfloat), triangle, GL_STATIC_DRAW);
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
        }
    }

    printf("\n");

    glfw_runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}


////////////////////////////////////////////////////////////////////////////////
//! Initialize GL with GLFW
////////////////////////////////////////////////////////////////////////////////
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    glViewport(0, 0, width, height);
}

bool glfw_initGL(int* argc, char** argv)
{
    if (!glfwInit())
        exit(EXIT_FAILURE);

    // Create window
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 4);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    window = glfwCreateWindow(window_width, window_height, "Cuda GL Interop(VBO)", NULL, NULL);
    if (NULL == window)
    {
        fprintf(stderr, "Failed to create GLFW window.\n");
        glfwTerminate();
        return EXIT_FAILURE;
    }
    glfwMakeContextCurrent(window);

    // Set callbacks
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
    glfwSetKeyCallback(window, glfw_keyboard);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
    glfwSetCursorPosCallback(window, glfw_cursor_pos_callback);
    glfwSetMouseButtonCallback(window, glfw_mouse);

    GLenum glerr = glewInit();
    if (GLEW_OK != glerr)
    {
        fprintf(stderr, "glewInit Error: %s\n", glewGetErrorString(glerr));
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    init_shaders();

    glerr = glGetError();

    return true;
}

bool glfw_runTest(int argc, char** argv, char* ref_file) {
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char**)argv);

    // command line mode only
    if (ref_file != NULL)
    {
        // create VBO
        checkCudaErrors(hipMalloc((void**)&d_vbo_buffer, mesh_width * mesh_height * 4 * sizeof(float)));

        // run the cuda part
        runAutoTest(devID, argv, ref_file);

        // check result of Cuda step
        checkResultCuda(argc, argv, vbo);

        hipFree(d_vbo_buffer);
        d_vbo_buffer = NULL;
    }
    else
    {
        // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
        if (false == glfw_initGL(&argc, argv))
        {
            return false;
        }

        // create VBO
        createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

        // run the cuda part
        runCuda(&cuda_vbo_resource);

        basic_triangle_vbo();

        // start rendering mainloop
        while (!glfwWindowShouldClose(window)) {
            glfw_display();
        }
    }

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

    launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
    printf("sdkDumpBin: <%s>\n", filename);
    FILE *fp;
    FOPEN(fp, filename, "wb");
    fwrite(data, bytes, 1, fp);
    fflush(fp);
    fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runAutoTest(int devID, char **argv, char *ref_file)
{
    char *reference_file = NULL;
    void *imageData = malloc(mesh_width*mesh_height*sizeof(float));

    // execute the kernel
    launch_kernel((float4 *)d_vbo_buffer, mesh_width, mesh_height, g_fAnim);

    hipDeviceSynchronize();
    getLastCudaError("launch_kernel failed");

    checkCudaErrors(hipMemcpy(imageData, d_vbo_buffer, mesh_width*mesh_height*sizeof(float), hipMemcpyDeviceToHost));

    sdkDumpBin2(imageData, mesh_width*mesh_height*sizeof(float), "simpleGL.bin");
    reference_file = sdkFindFilePath(ref_file, argv[0]);

    if (reference_file &&
        !sdkCompareBin2BinFloat("simpleGL.bin", reference_file,
                                mesh_width*mesh_height*sizeof(float),
                                MAX_EPSILON_ERROR, THRESHOLD, pArgv[0]))
    {
        g_TotalErrors++;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    //SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////

void glfw_display() {
    //sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClearColor(0.0f, 0.4f, 0.6f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    // Process input to update view matrix
    g_eyePosPolar = glm::vec3(translate_z, rotate_x, rotate_y);
    g_eyePosCartesian = glm::vec3(polarToCartesianPoint(g_eyePosPolar));
    g_viewMTX = glm::lookAt(g_eyePosCartesian, g_look_at_point, g_up_vector);
    
    // Set uniforms
    // MVP transforms
    glUniformMatrix4fv(glGetUniformLocation(g_ProgramID, "model"), GL_ONE, GL_FALSE, glm::value_ptr(g_modelMTX));
    glUniformMatrix4fv(glGetUniformLocation(g_ProgramID, "view"), GL_ONE, GL_FALSE, glm::value_ptr(g_viewMTX));
    glUniformMatrix4fv(glGetUniformLocation(g_ProgramID, "proj"), GL_ONE, GL_FALSE, glm::value_ptr(g_projMTX));

    // Bind vertex array (cuda interop)
    glBindVertexArray(tri_vao);
    glBindBuffer(GL_ARRAY_BUFFER, tri_vbo);

    GLuint loc = glGetAttribLocation(g_ProgramID, "vPosition");
    glEnableVertexAttribArray(loc);
    glVertexAttribPointer(loc, 3, GL_FLOAT, GL_FALSE, 7 * sizeof(GL_FLOAT), (void*)0);
    GLuint clr = glGetAttribLocation(g_ProgramID, "vColor");
    glEnableVertexAttribArray(clr);
    glVertexAttribPointer(clr, 4, GL_FLOAT, GL_FALSE, 7 * sizeof(GL_FLOAT), (void*)(3 * sizeof(float)));

    // Draw Arrays
    // Reset pointers??
    glDrawArrays(GL_TRIANGLES, 0, 3 * 7 * sizeof(GLfloat));
    //glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glFlush();
    
    glfwSwapBuffers(window);
    glfwPollEvents();

    g_fAnim += 0.01f;

    //sdkStopTimer(&timer);
    //computeFPS();
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////

void glfw_keyboard(GLFWwindow* window, int key, int scancode, int action, int mods) {
    switch (key)
    {
    case GLFW_KEY_ESCAPE:
        if (action == GLFW_PRESS) {
            glfwSetWindowShouldClose(window, true);
            printf("KEY: ESCAPE PRESSED => EXIT");
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////

void glfw_mouse(GLFWwindow* window, int button, int action, int mods) {
    if (action == GLFW_PRESS) {
        mouse_buttons |= 1 << button;
        glfwGetCursorPos(window, &glfw_mouse_old_x, &glfw_mouse_old_y);
    } 
    else if (action == GLFW_RELEASE) {
        mouse_buttons = 0;
    }
}

void glfw_cursor_pos_callback(GLFWwindow* window, double xpos, double ypos) {
    double dx, dy;
    dx = (double)(xpos - glfw_mouse_old_x);
    dy = (double)(ypos - glfw_mouse_old_y);

    if (mouse_buttons & (1 << GLFW_MOUSE_BUTTON_LEFT)) {
        rotate_x += dy * 0.02f;
        rotate_y += dx * 0.02f;
    }
    else if (mouse_buttons & (1 << GLFW_MOUSE_BUTTON_RIGHT)) {
        translate_z += dy * 0.001f;
    }

    mouse_old_x = xpos;
    mouse_old_y = ypos;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo)
{
    if (!d_vbo_buffer)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

        // map buffer object
        glBindBuffer(GL_ARRAY_BUFFER, vbo);
        float *data = (float *) glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

        // check result
        if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
        {
            // write file for regression test
            sdkWriteFile<float>("./data/regression.dat",
                                data, mesh_width * mesh_height * 3, 0.0, false);
        }

        // unmap GL buffer object
        if (!glUnmapBuffer(GL_ARRAY_BUFFER))
        {
            fprintf(stderr, "Unmap buffer failed.\n");
            fflush(stderr);
        }

        checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo,
                                                     cudaGraphicsMapFlagsWriteDiscard));

        //SDK_CHECK_ERROR_GL();
    }
}
